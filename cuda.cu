
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string>
#include <cmath>
#include <algorithm>
#include <chrono>

using namespace std;

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

// Forward declarations of CUDA kernels
__global__ void Rgb2GrayKernel(uchar3 * inPixels, int width, int height, uint8_t * outPixels);
__global__ void backwardEnergyKernel(uint8_t * inPixels, int width, int height, int* xfilter, int* yfilter, int * importants);
__global__ void carvingKernel(int *leastSignificantPixel, uchar3 *outPixels, uint8_t *grayPixels, int * importants, int width);
__global__ void seamsScoreKernel(int *importants, int *score, int width, int height, int fromRow);
__global__ void forwardEnergyKernel(uint8_t *grayPixels, int width, int height, float *energy);
__global__ void hybridEnergyKernel(int *backwardEnergy, float *forwardEnergy, float *hybridEnergy, int width, int height);
__global__ void convertToIntKernel(float *floatEnergy, int *intEnergy, int width, int height);
__global__ void updateLocalBackwardEnergyKernel(uint8_t *grayPixels, int *importants, int width, int height, int *seamPath);

// Add at the top of the file, after other declarations
__device__ __constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__device__ __constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

/**
 * Helper class for measuring CUDA execution time
 * Uses CUDA events for accurate timing of GPU operations
 * Methods:
 *   - Start(): Records start event
 *   - Stop(): Records stop event
 *   - Elapsed(): Returns elapsed time in milliseconds
 */
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);                                                                 
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
    FILE * f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);
    
    if (strcmp(type, "P3") != 0) 
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);
    
    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) 
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, int originalWidth, char *fileName)
{
    FILE * f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }   

    fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            int i = r * originalWidth + c;
            fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
        }
    }
    
    fclose(f);
}

__device__ int d_originalWidth;

/**
 * CUDA kernel for converting RGB image to grayscale
 * Each thread processes one pixel using the standard luminance formula
 * @param inPixels - Input RGB image in device memory
 * @param width - Image width
 * @param height - Image height
 * @param outPixels - Output grayscale image in device memory
 */
__global__ void Rgb2GrayKernel(uchar3 * inPixels, int width, int height, uint8_t * outPixels) {
    size_t r = blockIdx.y * blockDim.y + threadIdx.y;
    size_t c = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = r * width + c;
    if (r < height && c < width) {
        outPixels[i] = 0.299f * inPixels[i].x
                    + 0.587f * inPixels[i].y
                    + 0.114f * inPixels[i].z;
    }
}

/**
 * CUDA kernel for calculating pixel importance using Sobel edge detection
 * Each thread processes one pixel, applying 3x3 Sobel filters
 * @param inPixels - Grayscale image in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param xfilter - Sobel x-filter in device memory
 * @param yfilter - Sobel y-filter in device memory
 * @param importants - Output importance values in device memory
 */
__global__ void backwardEnergyKernel(uint8_t * inPixels, int width, int height, int* xfilter, int* yfilter, int * importants) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) 
    {
        return;
    }

    int x = 0, y = 0;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            // Handling the case of pixels falling outside the boundary
            int r = (py - 1) + i;
            int c = (px - 1) + j;
            if (r < 0) 
                r = 0;
            else if (r >= height) 
                r = height - 1;
        
            if (c < 0) {
                c = 0;
            } else if (c >= width) {
                c = width - 1;
            }
            int sobelIdx = i * 3 + j;
            uint8_t closest = inPixels[r * d_originalWidth + c];
            x += static_cast<int>(closest) * d_xSobel[sobelIdx];
            y += static_cast<int>(closest) * d_ySobel[sobelIdx];
        }
    }
    importants[py * d_originalWidth + px] = abs(x) + abs(y);
}

/**
 * CUDA kernel for removing a seam from the image
 * Each thread block processes one row of the image
 * @param leastSignificantPixel - Array of column indices for the seam
 * @param outPixels - Output image in device memory
 * @param grayPixels - Grayscale image in device memory
 * @param importants - Importance values in device memory
 * @param width - Current image width
 */
__global__ void carvingKernel(int *leastSignificantPixel, uchar3 *outPixels, uint8_t *grayPixels, int * importants, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_originalWidth;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        importants[baseIdx + i] = importants[baseIdx + i + 1];
    }
}

/**
 * Host function for tracing the optimal seam path
 * @param score - Cumulative scores from device memory
 * @param leastSignificantPixel - Output array for seam column indices
 * @param width - Current image width
 * @param height - Image height
 * @param originalWidth - Original image width
 */
void trace(int *score, int *leastSignificantPixel, int width, int height, int originalWidth) {
    int minCol = 0, r = height - 1;
    for (int c = 1; c < width; ++c) {
        if (score[r * originalWidth + c] < score[r * originalWidth + minCol])
            minCol = c;
    }
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * originalWidth + minCol;
            int min = score[aboveIdx], minColCpy = minCol;
            if (minColCpy > 0 && score[aboveIdx - 1] < min) {
                min = score[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && score[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

/**
 * CUDA kernel for dynamic programming seam finding
 * Uses shared memory for efficient data access
 * @param importants - Pixel importance values in device memory
 * @param score - Output cumulative scores in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param fromRow - Starting row for this kernel execution
 */
__global__ void seamsScoreKernel(int *importants, int *score, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x >> 1;

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x;

    if (fromRow == 0 && col >= 0 && col < width) {
        score[col] = importants[col];
    }
    __syncthreads();

    for (int stride = fromRow != 0 ? 0 : 1; stride < halfBlock && fromRow + stride < height; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow + stride;
            int curCol = col + stride;

            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_originalWidth + curCol;
                int aboveIdx = (curRow - 1) * d_originalWidth + curCol;

                int min = score[aboveIdx];
                if (curCol > 0 && score[aboveIdx - 1] < min) {
                    min = score[aboveIdx - 1];
                }
                if (curCol < width - 1 && score[aboveIdx + 1] < min) {
                    min = score[aboveIdx + 1];
                }

                score[idx] = min + importants[idx];
            }
        }
        __syncthreads();
    }
}

/**
 * CUDA kernel for calculating forward energy
 * Each thread processes one pixel
 * @param grayPixels - Grayscale image in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param energy - Output energy values in device memory
 */
__global__ void forwardEnergyKernel(uint8_t *grayPixels, int width, int height, float *energy) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    
    if (row == 0) {
        energy[idx] = 0.0f;
        return;
    }
    
    // Get neighboring pixel values with proper boundary handling like getClosest()
    int leftCol = max(0, col - 1);
    int rightCol = min(width - 1, col + 1);
    int upRow = max(0, row - 1);
    
    float left = static_cast<float>(grayPixels[row * d_originalWidth + leftCol]);
    float right = static_cast<float>(grayPixels[row * d_originalWidth + rightCol]);
    float up = static_cast<float>(grayPixels[upRow * d_originalWidth + col]);
    float upLeft = static_cast<float>(grayPixels[upRow * d_originalWidth + leftCol]);
    float upRight = static_cast<float>(grayPixels[upRow * d_originalWidth + rightCol]);
    
    // Compute directional costs using floating-point
    float cU = fabsf(right - left);  // Cost for going straight up
    float cL = cU + fabsf(up - left);  // Cost for going up-left
    float cR = cU + fabsf(up - right);  // Cost for going up-right
    
    // Get minimum previous path cost
    float min_energy = energy[idx - d_originalWidth] + cU;
    if (col > 0) {
        min_energy = fminf(min_energy, energy[idx - d_originalWidth - 1] + cL);
    }
    if (col < width - 1) {
        min_energy = fminf(min_energy, energy[idx - d_originalWidth + 1] + cR);
    }
    
    energy[idx] = min_energy;
}

/**
 * CUDA kernel for combining backward and forward energy
 * Each thread processes one pixel
 * @param backwardEnergy - Sobel-based energy values in device memory
 * @param forwardEnergy - Forward energy values in device memory
 * @param hybridEnergy - Output combined energy values in device memory
 * @param width - Current image width
 * @param height - Image height
 */
__global__ void hybridEnergyKernel(int *backwardEnergy, float *forwardEnergy, float *hybridEnergy, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    
    // Normalize both energy values to 0-1 range
    float backwardNorm = static_cast<float>(backwardEnergy[idx]) / 255.0f;
    float forwardNorm = forwardEnergy[idx] / 255.0f;  // forwardEnergy is already in float
    
    // Choose the higher energy value
    float hybridVal = fmaxf(backwardNorm, forwardNorm);
    
    // Store the normalized hybrid energy
    hybridEnergy[idx] = hybridVal;
}

/**
 * CUDA kernel for converting float energy values to integers
 * Each thread processes one pixel
 * @param floatEnergy - Input float energy values in device memory
 * @param intEnergy - Output integer energy values in device memory
 * @param width - Current image width
 * @param height - Image height
 */
__global__ void convertToIntKernel(float *floatEnergy, int *intEnergy, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    intEnergy[idx] = static_cast<int>(floatEnergy[idx] * 255.0f);
}

/**
 * CUDA kernel for updating backward energy locally after seam removal
 * Each thread updates one pixel in the affected window
 */
__global__ void updateLocalBackwardEnergyKernel(uint8_t *grayPixels, int *importants, int width, int height, int *seamPath) {
    int row = blockIdx.x;
    if (row >= height) return;
    
    int seamCol = seamPath[row];
    int baseIdx = row * d_originalWidth;
    
    // Update a 5-pixel window around the seam
    for (int dc = -2; dc <= 2; dc++) {
        int col = seamCol + dc;
        if (col >= 0 && col < width - 1) {  // width-1 because we already shifted pixels
            int idx = baseIdx + col;
            int x = 0, y = 0;
            
            // Apply Sobel filter with safe boundary handling
            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    int r = (row - 1) + i;
                    int c = (col - 1) + j;
                    
                    // Replicate border pixels like getClosest()
                    if (r < 0) r = 0;
                    else if (r >= height) r = height - 1;
                    
                    if (c < 0) c = 0;
                    else if (c >= width - 1) c = width - 2;  // width-2 because we already shifted
                    
                    int pixel = static_cast<int>(grayPixels[r * d_originalWidth + c]);
                    int sobelIdx = i * 3 + j;
                    x += pixel * d_xSobel[sobelIdx];
                    y += pixel * d_ySobel[sobelIdx];
                }
            }
            importants[idx] = abs(x) + abs(y);
        }
    }
}

/**
 * Main CUDA implementation of seam carving
 * Manages device memory and coordinates kernel execution
 * @param inPixels - Input RGB image
 * @param width - Initial image width
 * @param height - Image height
 * @param targetWidth - Desired final width
 * @param outPixels - Output image after seam removal
 * @param blockSize - CUDA block dimensions for kernel execution
 */
void seamCarvingByDevice(uchar3 *inPixels, int width, int height, int targetWidth, uchar3* outPixels, dim3 blockSize) {
    // Start total timer
    auto totalStart = std::chrono::high_resolution_clock::now();
    
    // Initialize timing variables
    double totalGrayscaleTime = 0.0;
    double totalBackwardEnergyTime = 0.0;
    double totalForwardEnergyTime = 0.0;
    double totalHybridEnergyTime = 0.0;
    double totalDpTime = 0.0;
    double totalSeamTracingTime = 0.0;

    // Memory allocation
    uchar3 *d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_importants;
    CHECK(hipMalloc(&d_importants, width * height * sizeof(int)));
    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));
    int * d_score;
    CHECK(hipMalloc(&d_score, width * height * sizeof(int)));
    float * d_forwardEnergy;
    CHECK(hipMalloc(&d_forwardEnergy, width * height * sizeof(float)));
    float * d_hybridEnergy;
    CHECK(hipMalloc(&d_hybridEnergy, width * height * sizeof(float)));

    int * importants = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int * score = (int *)malloc(width * height * sizeof(int));

    // Allocate to shared memory
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    
    // block size use to calculate seam score table
    int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_originalWidth), &width, sizeof(int)));
    const int originalWidth = width;

    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // Step 1: Convert RGB image to grayscale
    auto grayscaleStart = std::chrono::high_resolution_clock::now();
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    Rgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());
    auto grayscaleEnd = std::chrono::high_resolution_clock::now();
    totalGrayscaleTime = std::chrono::duration_cast<std::chrono::microseconds>(grayscaleEnd - grayscaleStart).count() / 1000.0;

    // Loop to delete each seam
    while (width > targetWidth) {
        // Step 2: Calculate backward energy
        auto backwardStart = std::chrono::high_resolution_clock::now();
        backwardEnergyKernel<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, nullptr, nullptr, d_importants);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto backwardEnd = std::chrono::high_resolution_clock::now();
        totalBackwardEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(backwardEnd - backwardStart).count() / 1000.0;

        // Step 2.1: Calculate forward energy
        auto forwardStart = std::chrono::high_resolution_clock::now();
        forwardEnergyKernel<<<gridSize, blockSize>>>(d_grayPixels, width, height, d_forwardEnergy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto forwardEnd = std::chrono::high_resolution_clock::now();
        totalForwardEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(forwardEnd - forwardStart).count() / 1000.0;

        // Step 2.2: Combine energies using hybrid approach
        auto hybridStart = std::chrono::high_resolution_clock::now();
        hybridEnergyKernel<<<gridSize, blockSize>>>(d_importants, d_forwardEnergy, d_hybridEnergy, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto hybridEnd = std::chrono::high_resolution_clock::now();
        totalHybridEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(hybridEnd - hybridStart).count() / 1000.0;

        // Convert hybrid energy to integer for seam finding
        dim3 convertGrid((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
        convertToIntKernel<<<convertGrid, blockSize>>>(d_hybridEnergy, d_importants, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // Step 3: Calculate the seam table to find the seam with the smallest value
        auto dpStart = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < height; i += (stripHeight >> 1)) {
            seamsScoreKernel<<<gridSizeDp, blockSizeDp>>>(d_importants, d_score, width, height, i);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }
        auto dpEnd = std::chrono::high_resolution_clock::now();
        totalDpTime += std::chrono::duration_cast<std::chrono::microseconds>(dpEnd - dpStart).count() / 1000.0;

        // From the bottom smallest pixel, trace up the first line to find the seam.
        auto seamTracingStart = std::chrono::high_resolution_clock::now();
        CHECK(hipMemcpy(score, d_score, originalWidth * height * sizeof(int), hipMemcpyDeviceToHost));
        trace(score, leastSignificantPixel, width, height, originalWidth);
        CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        
        // Step 4: Delete the seam and update local importance values
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_importants, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        // Update local importance values around the removed seam
        updateLocalBackwardEnergyKernel<<<height, 1>>>(d_grayPixels, d_importants, width, height, d_leastSignificantPixel);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        auto seamTracingEnd = std::chrono::high_resolution_clock::now();
        totalSeamTracingTime += std::chrono::duration_cast<std::chrono::microseconds>(seamTracingEnd - seamTracingStart).count() / 1000.0;

        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, originalWidth * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_importants));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_score));
    CHECK(hipFree(d_forwardEnergy));
    CHECK(hipFree(d_hybridEnergy));
    free(score);
    free(leastSignificantPixel);
    free(importants);

    // Calculate total time
    auto totalEnd = std::chrono::high_resolution_clock::now();
    double totalTime = std::chrono::duration_cast<std::chrono::microseconds>(totalEnd - totalStart).count() / 1000.0;

    // Print performance metrics
    printf("\nCUDA Implementation Performance Analysis:\n");
    printf("---------------------------------\n");
    printf("Grayscale conversion: %.2f ms\n", totalGrayscaleTime);
    printf("Backward energy (Sobel): %.2f ms\n", totalBackwardEnergyTime);
    printf("Forward energy: %.2f ms\n", totalForwardEnergyTime);
    printf("Hybrid energy: %.2f ms\n", totalHybridEnergyTime);
    printf("Dynamic programming: %.2f ms\n", totalDpTime);
    printf("Seam tracing and removal: %.2f ms\n", totalSeamTracingTime);
    printf("---------------------------------\n");
    printf("Total seam carving time: %.2f ms\n\n", totalTime);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
    float err = 0;
    for (int i = 0; i < n; i++)
    {
        err += abs((int)a1[i].x - (int)a2[i].x);
        err += abs((int)a1[i].y - (int)a2[i].y);
        err += abs((int)a1[i].z - (int)a2[i].z);
    }
    err /= (n * 3);
    return err;
}

/**
 * Concatenates two strings
 * @param s1 - First string
 * @param s2 - Second string
 * @return Concatenated string
 */
char *concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n\n");

}

int main(int argc, char ** argv)
{   
    if (argc != 4 && argc != 6)
    {
        printf("The number of arguments is invalid\n");
        return EXIT_FAILURE;
    }

    printDeviceInfo();

    // Read input RGB image file
    int width, height;
    uchar3 *inPixels;
    readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    int numSeamRemoved = stoi(argv[3]);
    if (numSeamRemoved <= 0 || numSeamRemoved >= width)
        return EXIT_FAILURE; // invalid ratio
    printf("Number of seam removed: %d\n\n", numSeamRemoved);

    int targetWidth = width - numSeamRemoved;

    // seam carving using device
    uchar3 * outPixels= (uchar3 *)malloc(width * height * sizeof(uchar3));
    dim3 blockSize(16, 16); // Default
    if (argc == 6)
    {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    } 
    seamCarvingByDevice(inPixels, width, height, targetWidth, outPixels, blockSize);
    printf("Image size after seam carving (new_width x height): %i x %i\n\n", targetWidth, height);
    
    // Write results to files
    char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension
    writePnm(outPixels, targetWidth, height, width, concatStr(outFileNameBase, "_cuda.pnm"));

    // Free memories
    free(inPixels);
    // free(correctOutPixels);
    free(outPixels);
}