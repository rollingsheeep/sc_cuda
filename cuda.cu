
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string>
#include <cmath>
#include <algorithm>
#include <chrono>

using namespace std;

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

// Forward declarations of CUDA kernels
__global__ void Rgb2GrayKernel(uchar3 * inPixels, int width, int height, uint8_t * outPixels);
__global__ void pixelsImportantKernel(uint8_t * inPixels, int width, int height, int* xfilter, int* yfilter, int * importants);
__global__ void carvingKernel(int *leastSignificantPixel, uchar3 *outPixels, uint8_t *grayPixels, int * importants, int width);
__global__ void seamsScoreKernel(int *importants, int *score, int width, int height, int fromRow);
__global__ void forwardEnergyKernel(uint8_t *grayPixels, int width, int height, float *energy);
__global__ void hybridEnergyKernel(int *backwardEnergy, float *forwardEnergy, float *hybridEnergy, int width, int height);
__global__ void convertToIntKernel(float *floatEnergy, int *intEnergy, int width, int height);

/**
 * Helper class for measuring CUDA execution time
 * Uses CUDA events for accurate timing of GPU operations
 * Methods:
 *   - Start(): Records start event
 *   - Stop(): Records stop event
 *   - Elapsed(): Returns elapsed time in milliseconds
 */
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);                                                                 
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
    FILE * f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);
    
    if (strcmp(type, "P3") != 0) 
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);
    
    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) 
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, int originalWidth, char *fileName)
{
    FILE * f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }   

    fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            int i = r * originalWidth + c;
            fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
        }
    }
    
    fclose(f);
}

int xSobel[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
int ySobel[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};

__device__ int d_originalWidth;

/**
 * CUDA kernel for converting RGB image to grayscale
 * Each thread processes one pixel using the standard luminance formula
 * @param inPixels - Input RGB image in device memory
 * @param width - Image width
 * @param height - Image height
 * @param outPixels - Output grayscale image in device memory
 */
__global__ void Rgb2GrayKernel(uchar3 * inPixels, int width, int height, uint8_t * outPixels) {
    size_t r = blockIdx.y * blockDim.y + threadIdx.y;
    size_t c = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = r * width + c;
    if (r < height && c < width) {
        outPixels[i] = 0.299f * inPixels[i].x
                    + 0.587f * inPixels[i].y
                    + 0.114f * inPixels[i].z;
    }
}

/**
 * CUDA kernel for calculating pixel importance using Sobel edge detection
 * Each thread processes one pixel, applying 3x3 Sobel filters
 * @param inPixels - Grayscale image in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param xfilter - Sobel x-filter in device memory
 * @param yfilter - Sobel y-filter in device memory
 * @param importants - Output importance values in device memory
 */
__global__ void pixelsImportantKernel(uint8_t * inPixels, int width, int height, int* xfilter, int* yfilter, int * importants) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	if (px >= width || py >= height) 
	{
		return;
	}

    int x = 0, y = 0;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {

            // Handling the case of pixels falling outside the boundary
            int r = (py - 1) + i;
            int c = (px - 1) + j;
            if (r < 0) 
                r = 0;
            else if (r >= height) 
                r = height - 1;
        
            if (c < 0) {
                c = 0;
            } else if (c >= width) {
                c = width - 1;
            }
            int idx = i * 3 + j;
            uint8_t closest = inPixels[r * d_originalWidth + c];
            x += closest * xfilter[idx];
            y += closest * yfilter[idx];
        }
    }
    importants[py * d_originalWidth + px] = abs(x) + abs(y);
}

/**
 * CUDA kernel for removing a seam from the image
 * Each thread block processes one row of the image
 * @param leastSignificantPixel - Array of column indices for the seam
 * @param outPixels - Output image in device memory
 * @param grayPixels - Grayscale image in device memory
 * @param importants - Importance values in device memory
 * @param width - Current image width
 */
__global__ void carvingKernel(int *leastSignificantPixel, uchar3 *outPixels, uint8_t *grayPixels, int * importants, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_originalWidth;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        importants[baseIdx + i] = importants[baseIdx + i + 1];
    }
}

/**
 * Host function for tracing the optimal seam path
 * @param score - Cumulative scores from device memory
 * @param leastSignificantPixel - Output array for seam column indices
 * @param width - Current image width
 * @param height - Image height
 * @param originalWidth - Original image width
 */
void trace(int *score, int *leastSignificantPixel, int width, int height, int originalWidth) {
    int minCol = 0, r = height - 1;
    for (int c = 1; c < width; ++c) {
        if (score[r * originalWidth + c] < score[r * originalWidth + minCol])
            minCol = c;
    }
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * originalWidth + minCol;
            int min = score[aboveIdx], minColCpy = minCol;
            if (minColCpy > 0 && score[aboveIdx - 1] < min) {
                min = score[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && score[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

/**
 * CUDA kernel for dynamic programming seam finding
 * Uses shared memory for efficient data access
 * @param importants - Pixel importance values in device memory
 * @param score - Output cumulative scores in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param fromRow - Starting row for this kernel execution
 */
__global__ void seamsScoreKernel(int *importants, int *score, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x >> 1;

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x;

    if (fromRow == 0 && col >= 0 && col < width) {
        score[col] = importants[col];
    }
    __syncthreads();

    for (int stride = fromRow != 0 ? 0 : 1; stride < halfBlock && fromRow + stride < height; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow + stride;
            int curCol = col + stride;

            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_originalWidth + curCol;
                int aboveIdx = (curRow - 1) * d_originalWidth + curCol;

                int min = score[aboveIdx];
                if (curCol > 0 && score[aboveIdx - 1] < min) {
                    min = score[aboveIdx - 1];
                }
                if (curCol < width - 1 && score[aboveIdx + 1] < min) {
                    min = score[aboveIdx + 1];
                }

                score[idx] = min + importants[idx];
            }
        }
        __syncthreads();
    }
}

/**
 * CUDA kernel for calculating forward energy
 * Each thread processes one pixel
 * @param grayPixels - Grayscale image in device memory
 * @param width - Current image width
 * @param height - Image height
 * @param energy - Output energy values in device memory
 */
__global__ void forwardEnergyKernel(uint8_t *grayPixels, int width, int height, float *energy) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    
    if (row == 0) {
        energy[idx] = 0.0f;
        return;
    }
    
    // Get neighboring pixel values safely with bounds
    float left = (col > 0) ? static_cast<float>(grayPixels[idx - 1]) : static_cast<float>(grayPixels[idx]);
    float right = (col < width - 1) ? static_cast<float>(grayPixels[idx + 1]) : static_cast<float>(grayPixels[idx]);
    float up = static_cast<float>(grayPixels[idx - d_originalWidth]);
    float upLeft = (col > 0) ? static_cast<float>(grayPixels[idx - d_originalWidth - 1]) : up;
    float upRight = (col < width - 1) ? static_cast<float>(grayPixels[idx - d_originalWidth + 1]) : up;
    
    // Compute directional costs using floating-point
    float cU = fabsf(right - left);  // Cost for going straight up
    float cL = cU + fabsf(up - left);  // Cost for going up-left
    float cR = cU + fabsf(up - right);  // Cost for going up-right
    
    // Get minimum previous path cost
    float min_energy = energy[idx - d_originalWidth] + cU;
    if (col > 0) {
        min_energy = fminf(min_energy, energy[idx - d_originalWidth - 1] + cL);
    }
    if (col < width - 1) {
        min_energy = fminf(min_energy, energy[idx - d_originalWidth + 1] + cR);
    }
    
    energy[idx] = min_energy;
}

/**
 * CUDA kernel for combining backward and forward energy
 * Each thread processes one pixel
 * @param backwardEnergy - Sobel-based energy values in device memory
 * @param forwardEnergy - Forward energy values in device memory
 * @param hybridEnergy - Output combined energy values in device memory
 * @param width - Current image width
 * @param height - Image height
 */
__global__ void hybridEnergyKernel(int *backwardEnergy, float *forwardEnergy, float *hybridEnergy, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    
    // Normalize both energy values to 0-1 range
    float backwardNorm = static_cast<float>(backwardEnergy[idx]) / 255.0f;
    float forwardNorm = forwardEnergy[idx] / 255.0f;  // forwardEnergy is already in float
    
    // Choose the higher energy value
    float hybridVal = fmaxf(backwardNorm, forwardNorm);
    
    // Store the normalized hybrid energy
    hybridEnergy[idx] = hybridVal;
}

/**
 * CUDA kernel for converting float energy values to integers
 * Each thread processes one pixel
 * @param floatEnergy - Input float energy values in device memory
 * @param intEnergy - Output integer energy values in device memory
 * @param width - Current image width
 * @param height - Image height
 */
__global__ void convertToIntKernel(float *floatEnergy, int *intEnergy, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col >= width || row >= height) return;
    
    int idx = row * d_originalWidth + col;
    intEnergy[idx] = static_cast<int>(floatEnergy[idx] * 255.0f);
}

/**
 * Main CUDA implementation of seam carving
 * Manages device memory and coordinates kernel execution
 * @param inPixels - Input RGB image
 * @param width - Initial image width
 * @param height - Image height
 * @param targetWidth - Desired final width
 * @param outPixels - Output image after seam removal
 * @param blockSize - CUDA block dimensions for kernel execution
 */
void seamCarvingByDevice(uchar3 *inPixels, int width, int height, int targetWidth, uchar3* outPixels, dim3 blockSize) {
    // Start total timer
    auto totalStart = std::chrono::high_resolution_clock::now();
    
    // Initialize timing variables
    double totalGrayscaleTime = 0.0;
    double totalBackwardEnergyTime = 0.0;
    double totalForwardEnergyTime = 0.0;
    double totalHybridEnergyTime = 0.0;
    double totalDpTime = 0.0;
    double totalSeamTracingTime = 0.0;
    double totalLocalUpdateTime = 0.0;

    // Memory allocation
    uchar3 *d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_importants;
    CHECK(hipMalloc(&d_importants, width * height * sizeof(int)));
    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));
    int * d_score;
    CHECK(hipMalloc(&d_score, width * height * sizeof(int)));
    float * d_forwardEnergy;
    CHECK(hipMalloc(&d_forwardEnergy, width * height * sizeof(float)));
    float * d_hybridEnergy;
    CHECK(hipMalloc(&d_hybridEnergy, width * height * sizeof(float)));

    int * importants = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int * score = (int *)malloc(width * height * sizeof(int));

    // Allocate to shared memory
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    
    // block size use to calculate seam score table
    int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_originalWidth), &width, sizeof(int)));
    const int originalWidth = width;

    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // Step 1: Convert RGB image to grayscale
    auto grayscaleStart = std::chrono::high_resolution_clock::now();
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    Rgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());
    auto grayscaleEnd = std::chrono::high_resolution_clock::now();
    totalGrayscaleTime = std::chrono::duration_cast<std::chrono::microseconds>(grayscaleEnd - grayscaleStart).count() / 1000.0;

    // Allocate and initialize values for 2 filters
    int _xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
    int _ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
    int* xfilter, *yfilter;
    int xysize = 9*sizeof(int);
    CHECK(hipMalloc(&xfilter, xysize))
    CHECK(hipMalloc(&yfilter, xysize))

    CHECK(hipMemcpy(xfilter, _xSobel, xysize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(yfilter, _ySobel, xysize, hipMemcpyHostToDevice));

    // Loop to delete each seam
    while (width > targetWidth) {
        // Step 2: Calculate backward energy
        auto backwardStart = std::chrono::high_resolution_clock::now();
        pixelsImportantKernel<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, xfilter, yfilter, d_importants);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto backwardEnd = std::chrono::high_resolution_clock::now();
        totalBackwardEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(backwardEnd - backwardStart).count() / 1000.0;

        // Step 2.1: Calculate forward energy
        auto forwardStart = std::chrono::high_resolution_clock::now();
        forwardEnergyKernel<<<gridSize, blockSize>>>(d_grayPixels, width, height, d_forwardEnergy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto forwardEnd = std::chrono::high_resolution_clock::now();
        totalForwardEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(forwardEnd - forwardStart).count() / 1000.0;

        // Step 2.2: Combine energies using hybrid approach
        auto hybridStart = std::chrono::high_resolution_clock::now();
        hybridEnergyKernel<<<gridSize, blockSize>>>(d_importants, d_forwardEnergy, d_hybridEnergy, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto hybridEnd = std::chrono::high_resolution_clock::now();
        totalHybridEnergyTime += std::chrono::duration_cast<std::chrono::microseconds>(hybridEnd - hybridStart).count() / 1000.0;

        // Convert hybrid energy to integer for seam finding
        dim3 convertGrid((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
        convertToIntKernel<<<convertGrid, blockSize>>>(d_hybridEnergy, d_importants, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // Step 3: Calculate the seam table to find the seam with the smallest value
        auto dpStart = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < height; i += (stripHeight >> 1)) {
            seamsScoreKernel<<<gridSizeDp, blockSizeDp>>>(d_importants, d_score, width, height, i);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }
        auto dpEnd = std::chrono::high_resolution_clock::now();
        totalDpTime += std::chrono::duration_cast<std::chrono::microseconds>(dpEnd - dpStart).count() / 1000.0;

        // From the bottom smallest pixel, trace up the first line to find the seam.
        auto seamTracingStart = std::chrono::high_resolution_clock::now();
        CHECK(hipMemcpy(score, d_score, originalWidth * height * sizeof(int), hipMemcpyDeviceToHost));
        trace(score, leastSignificantPixel, width, height, originalWidth);
        auto seamTracingEnd = std::chrono::high_resolution_clock::now();
        totalSeamTracingTime += std::chrono::duration_cast<std::chrono::microseconds>(seamTracingEnd - seamTracingStart).count() / 1000.0;

        // Step 4: Delete the seam found
        auto localUpdateStart = std::chrono::high_resolution_clock::now();
        CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_importants, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        auto localUpdateEnd = std::chrono::high_resolution_clock::now();
        totalLocalUpdateTime += std::chrono::duration_cast<std::chrono::microseconds>(localUpdateEnd - localUpdateStart).count() / 1000.0;
        
        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, originalWidth * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_importants));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_score));
    CHECK(hipFree(d_forwardEnergy));
    CHECK(hipFree(d_hybridEnergy));
    CHECK(hipFree(xfilter));
    CHECK(hipFree(yfilter));
    free(score);
    free(leastSignificantPixel);
    free(importants);

    // Calculate total time
    auto totalEnd = std::chrono::high_resolution_clock::now();
    double totalTime = std::chrono::duration_cast<std::chrono::microseconds>(totalEnd - totalStart).count() / 1000.0;

    // Print performance metrics
    printf("\nCUDA Implementation Performance Analysis:\n");
    printf("---------------------------------\n");
    printf("Grayscale conversion: %.2f ms\n", totalGrayscaleTime);
    printf("Backward energy (Sobel): %.2f ms\n", totalBackwardEnergyTime);
    printf("Forward energy: %.2f ms\n", totalForwardEnergyTime);
    printf("Hybrid energy: %.2f ms\n", totalHybridEnergyTime);
    printf("Dynamic programming: %.2f ms\n", totalDpTime);
    printf("Seam tracing and removal: %.2f ms\n", totalSeamTracingTime);
    printf("Local importance map updates: %.2f ms\n", totalLocalUpdateTime);
    printf("---------------------------------\n");
    printf("Total seam carving time: %.2f ms\n\n", totalTime);
}

// uint8_t getClosest(uint8_t *pixels, int r, int c, int width, int height, int originalWidth)
// {
//     if (r < 0) {
//         r = 0;
//     } else if (r >= height) {
//         r = height - 1;
//     }

//     if (c < 0) {
//         c = 0;
//     } else if (c >= width) {
//         c = width - 1;
//     }

//     return pixels[r * originalWidth + c];
// }

// int pixelsImportant(uint8_t * grayPixels, int row, int col, int width, int height, int originalWidth) {
//     int x = 0, y = 0;
//     for (int i = 0; i < 3; ++i) {
//         for (int j = 0; j < 3; ++j) {
//             uint8_t closest = getClosest(grayPixels, row - 1 + i, col - 1 + j, width, height, originalWidth);
//             x += closest * xSobel[i][j];
//             y += closest * ySobel[i][j];
//         }
//     }
//     return abs(x) + abs(y);
// }

// void RGB2Gray(uchar3 * inPixels, int width, int height, uint8_t * outPixels) {
//     for (int r = 0; r < height; ++r) {
//         for (int c = 0; c < width; ++c) {
//             int i = r * width + c;
//             outPixels[i] = 0.299f * inPixels[i].x + 0.587f * inPixels[i].y + 0.114f * inPixels[i].z;
//         }
//     }
// }

// void seamsScore(int *importants, int *score, int width, int height, int originalWidth) {
//     for (int c = 0; c < width; ++c) {
//         score[c] = importants[c];
//     }
//     for (int r = 1; r < height; ++r) {
//         for (int c = 0; c < width; ++c) {
//             int idx = r * originalWidth + c;
//             int aboveIdx = (r - 1) * originalWidth + c;

//             int min = score[aboveIdx];
//             if (c > 0 && score[aboveIdx - 1] < min) {
//                 min = score[aboveIdx - 1];
//             }
//             if (c < width - 1 && score[aboveIdx + 1] < min) {
//                 min = score[aboveIdx + 1];
//             }

//             score[idx] = min + importants[idx];
//         }
//     }
// }

// void seamCarvingByHost(uchar3 *inPixels, int width, int height, int targetWidth, uchar3* outPixels) {
//     GpuTimer timer;
//     timer.Start();

//     memcpy(outPixels, inPixels, width * height * sizeof(uchar3));

//     const int originalWidth = width;

//     // allocate memory
//     int *importants = (int *)malloc(width * height * sizeof(int));
//     int *score = (int *)malloc(width * height * sizeof(int));
//     uint8_t *grayPixels= (uint8_t *)malloc(width * height * sizeof(uint8_t));
    
//     // Convert to grayscale image
//     RGB2Gray(inPixels, width, height, grayPixels);

//     // Calculate pixel importance
//     for (int r = 0; r < height; ++r) {
//         for (int c = 0; c < width; ++c) {
//             importants[r * originalWidth + c] = pixelsImportant(grayPixels, r, c, width, height, width);
//         }
//     }

//     while (width > targetWidth) {
//         seamsScore(importants, score, width, height, originalWidth);

//         // find where seam starts
//         int minCol = 0, r = height - 1, prevMinCol;
//         for (int c = 1; c < width; ++c) {
//             if (score[r * originalWidth + c] < score[r * originalWidth + minCol])
//                 minCol = c;
//         }

//         // trace and remove seams
//         for (; r >= 0; --r) {
//             for (int i = minCol; i < width - 1; ++i) {
//                 outPixels[r * originalWidth + i] = outPixels[r * originalWidth + i + 1];
//                 grayPixels[r * originalWidth + i] = grayPixels[r * originalWidth + i + 1];
//                 importants[r * originalWidth + i] = importants[r * originalWidth + i + 1];
//             }

            
//             if (r < height - 1) {
//                 for (int affectedCol = max(0, prevMinCol - 2); affectedCol <= prevMinCol + 2 && affectedCol < width - 1; ++affectedCol) {
//                     importants[(r + 1) * originalWidth + affectedCol] = pixelsImportant(grayPixels, r + 1, affectedCol, width - 1, height, originalWidth);
//                 }
//             }

            
//             if (r > 0) {
//                 prevMinCol = minCol;

//                 int aboveIdx = (r - 1) * originalWidth + minCol;
//                 int min = score[aboveIdx], minColCpy = minCol;
//                 if (minColCpy > 0 && score[aboveIdx - 1] < min) {
//                     min = score[aboveIdx - 1];
//                     minCol = minColCpy - 1;
//                 }
//                 if (minColCpy < width - 1 && score[aboveIdx + 1] < min) {
//                     minCol = minColCpy + 1;
//                 }
//             }
//         }

//         for (int affectedCol = max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
//             importants[affectedCol] = pixelsImportant(grayPixels, 0, affectedCol, width - 1, height, originalWidth);
//         }

//         --width;
//     }
    
//     free(grayPixels);
//     free(score);
//     free(importants);

//     timer.Stop();
//     float time = timer.Elapsed();
//     printf("Processing time (use host): %f ms\n\n", time);
// }

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
    float err = 0;
    for (int i = 0; i < n; i++)
    {
        err += abs((int)a1[i].x - (int)a2[i].x);
        err += abs((int)a1[i].y - (int)a2[i].y);
        err += abs((int)a1[i].z - (int)a2[i].z);
    }
    err /= (n * 3);
    return err;
}

/**
 * Concatenates two strings
 * @param s1 - First string
 * @param s2 - Second string
 * @return Concatenated string
 */
char *concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n\n");

}

int main(int argc, char ** argv)
{   
    if (argc != 4 && argc != 6)
    {
        printf("The number of arguments is invalid\n");
        return EXIT_FAILURE;
    }

    printDeviceInfo();

    // Read input RGB image file
    int width, height;
    uchar3 *inPixels;
    readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    int numSeamRemoved = stoi(argv[3]);
    if (numSeamRemoved <= 0 || numSeamRemoved >= width)
        return EXIT_FAILURE; // invalid ratio
    printf("Number of seam removed: %d\n\n", numSeamRemoved);

    int targetWidth = width - numSeamRemoved;

    // seam carving using host
    // uchar3 * correctOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    // seamCarvingByHost(inPixels, width, height, targetWidth, correctOutPixels);

    // seam carving using device
    uchar3 * outPixels= (uchar3 *)malloc(width * height * sizeof(uchar3));
    dim3 blockSize(16, 16); // Default
    if (argc == 6)
    {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    } 
    seamCarvingByDevice(inPixels, width, height, targetWidth, outPixels, blockSize);
    printf("Image size after seam carving (new_width x height): %i x %i\n\n", targetWidth, height);
    // Compute mean absolute error between host result and device result
    // float err = computeError(outPixels, correctOutPixels, width * height);
    // printf("Error between device result and host result: %f\n", err);
    
    // Write results to files
    char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension
    // writePnm(correctOutPixels, targetWidth, height, width, concatStr(outFileNameBase, "_host.pnm"));
    writePnm(outPixels, targetWidth, height, width, concatStr(outFileNameBase, "_device.pnm"));

    // Free memories
    free(inPixels);
    // free(correctOutPixels);
    free(outPixels);
}